#include "hip/hip_runtime.h"
#include "preprocess.h"
#include "cuda_utils.h"
#include <>
#include <opencv2/opencv.hpp>
#include <cmath>
#include <iostream>

// Static buffers for image data
static uint8_t* img_buffer_host = nullptr;
static uint8_t* img_buffer_device = nullptr;

// Affine transformation matrix struct
struct AffineMatrix {
    float value[6];
};

// CUDA kernel for warp affine transformation
__global__ void warpaffine_kernel(
    uint8_t* src, int src_line_size, int src_width, int src_height,
    float* dst, int dst_width, int dst_height, uint8_t const_value_st,
    AffineMatrix d2s, int edge) {
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= edge) return;

    // Extract transformation matrix values
    float m_x1 = d2s.value[0];
    float m_y1 = d2s.value[1];
    float m_z1 = d2s.value[2];
    float m_x2 = d2s.value[3];
    float m_y2 = d2s.value[4];
    float m_z2 = d2s.value[5];

    // Compute destination pixel coordinates
    int dx = position % dst_width;
    int dy = position / dst_width;
    float src_x = m_x1 * dx + m_y1 * dy + m_z1 + 0.5f;
    float src_y = m_x2 * dx + m_y2 * dy + m_z2 + 0.5f;
    float c0, c1, c2;

    // Check if source coordinates are out of bounds
    if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
        c0 = const_value_st;
        c1 = const_value_st;
        c2 = const_value_st;
    } else {
        // Bilinear interpolation
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t* v1 = const_value;
        uint8_t* v2 = const_value;
        uint8_t* v3 = const_value;
        uint8_t* v4 = const_value;

        if (y_low >= 0) {
            if (x_low >= 0)
                v1 = src + y_low * src_line_size + x_low * 3;
            if (x_high < src_width)
                v2 = src + y_low * src_line_size + x_high * 3;
        }
        if (y_high < src_height) {
            if (x_low >= 0)
                v3 = src + y_high * src_line_size + x_low * 3;
            if (x_high < src_width)
                v4 = src + y_high * src_line_size + x_high * 3;
        }

        c0 = w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0];
        c1 = w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1];
        c2 = w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2];
    }

    // BGR to RGB conversion
    float t = c2;
    c2 = c0;
    c0 = t;

    // Normalization to [0, 1]
    c0 = c0 / 255.0f;
    c1 = c1 / 255.0f;
    c2 = c2 / 255.0f;

    
    // Store in channel-separated format (rrrgggbbb) NCHW opencv stores in NHWC (rgbrgbrgb) tensorrt expects NCHW
    int area = dst_width * dst_height;
    float* pdst_c0 = dst + dy * dst_width + dx;
    float* pdst_c1 = pdst_c0 + area;
    float* pdst_c2 = pdst_c1 + area;
    *pdst_c0 = c0;
    *pdst_c1 = c1;
    *pdst_c2 = c2;
}

// Preprocessing function
void cuda_preprocess(
    uint8_t* src, int src_width, int src_height,
    float* dst, int dst_width, int dst_height,
    hipStream_t stream) {
    int img_size = src_width * src_height * 3;

    // Copy source image to pinned host memory
    memcpy(img_buffer_host, src, img_size);

    // Transfer imgage to device
    CUDA_CHECK(hipMemcpyAsync(img_buffer_device, img_buffer_host, img_size, 
                               hipMemcpyHostToDevice, stream));

    // Compute affine transformation matrices
    AffineMatrix s2d, d2s;
    float scale = std::min(static_cast<float>(dst_height) / src_height, 
                           static_cast<float>(dst_width) / src_width);


    s2d.value[0] = scale;
    s2d.value[1] = 0;
    s2d.value[2] = -scale * src_width * 0.5f + dst_width * 0.5f;
    s2d.value[3] = 0;
    s2d.value[4] = scale;
    s2d.value[5] = -scale * src_height * 0.5f + dst_height * 0.5f;

    cv::Mat m2x3_s2d(2, 3, CV_32F, s2d.value);

    cv::Mat m2x3_d2s(2, 3, CV_32F, d2s.value);
    
    cv::invertAffineTransform(m2x3_s2d, m2x3_d2s);
    memcpy(d2s.value, m2x3_d2s.ptr<float>(0), sizeof(d2s.value));

    // Launch kernel
    int jobs = dst_height * dst_width;
    int threads = 256;
    int blocks = ceil(jobs / static_cast<float>(threads));
    warpaffine_kernel<<<blocks, threads, 0, stream>>>(
        img_buffer_device, src_width * 3, src_width, src_height,
        dst, dst_width, dst_height, 128, d2s, jobs);

    CUDA_CHECK(hipGetLastError()); // Check for kernel launch errors
}

// Initialize preprocessing buffers
void cuda_preprocess_init(int max_image_size) {
    CUDA_CHECK(hipHostMalloc((void**)&img_buffer_host, max_image_size * 3));
    CUDA_CHECK(hipMalloc((void**)&img_buffer_device, max_image_size * 3));
}

// Cleanup preprocessing buffers
void cuda_preprocess_destroy() {
    CUDA_CHECK(hipFree(img_buffer_device));
    CUDA_CHECK(hipHostFree(img_buffer_host));
    img_buffer_device = nullptr;
    img_buffer_host = nullptr;
}